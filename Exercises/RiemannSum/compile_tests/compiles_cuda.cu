
#include <hip/hip_runtime.h>
#include <cstdlib>

__global__
void kernel(int sz, double* data)
{
    int _beg = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = _beg; i < sz; ++i)
        data[i] += static_cast<double>(i);
}

int main()
{
    double* data = NULL;
    int blocks = 64;
    int grids = 64;
    int ret = hipMalloc(&data, blocks * grids * sizeof(double));
    if(ret != hipSuccess)
        return EXIT_FAILURE;
    kernel<<<grids, blocks>>>(blocks * grids, data);
    hipDeviceSynchronize();
    return EXIT_SUCCESS;
}
